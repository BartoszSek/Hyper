#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void matrixVectorMult(float *matrix, float *vector, float *result,int N,int k)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int idy = blockIdx.y * blockDim.y + threadIdx.y;
    if (idx < N && idy<N) {
        
        result[0] = matrix[idx*k]*vector[idy];
    }
}

int main(void)
{
    float elapsed_time;
    int nstreams = 32;
    // Initialize matrix and vector
    float matrix[nstreams*nstreams];
    float vector[nstreams];
    for (int i = 0; i < nstreams * nstreams; i++) {
        matrix[i] = (float)(rand() % 10);
    }
    for (int i = 0; i < nstreams; i++) {
        vector[i] = (float)(rand() % 10);
    }

    // Allocate memory on device
    float *d_matrix, *d_vector, *d_result;
    hipMalloc((void **)&d_matrix, nstreams * nstreams * sizeof(float));
    hipMalloc((void **)&d_vector, nstreams * sizeof(float));
    hipMalloc((void **)&d_result, nstreams * sizeof(float));

    // Copy data from host to device
    hipMemcpy(d_matrix, matrix, nstreams * nstreams * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vector, vector, nstreams * sizeof(float), hipMemcpyHostToDevice);

    // Allocate and initialize an array of stream handles
    hipStream_t *streams =(hipStream_t *)malloc(nstreams * sizeof(hipStream_t));

    for (int i = 0; i < nstreams; i++) {
    hipStreamCreate(&(streams[i]));
    }

    hipEvent_t start_event, stop_event;
    hipEventCreate(&start_event);
    hipEventCreate(&stop_event);
    hipEventRecord(start_event, 0);
    // Launch kernel
    dim3 block_size(16, 16, 1);
    dim3 grid_size((nstreams + block_size.x - 1) / block_size.x,(nstreams + block_size.y - 1) / block_size.y , 1);
    
    for(int i = 0; i < nstreams; ++i){
    matrixVectorMult<<<grid_size, block_size,0,streams[i]>>>(d_matrix, d_vector, &d_result[i],nstreams,i);
    }
    hipEventRecord(stop_event, 0);
    hipEventSynchronize(stop_event);
    hipEventElapsedTime(&elapsed_time,start_event,stop_event);
    printf("Measured time for sample = %.3fus\n", elapsed_time);
    
    // Copy data from device to host
    float result[nstreams];
    hipMemcpy(result, d_result, nstreams * sizeof(float), hipMemcpyDeviceToHost);

    // Print results
//    printf("Matrix:\n");
//    for (int i = 0; i < nstreams; i++) {
//        for (int j = 0; j < nstreams; j++) {
//            printf("%f ", matrix[i * nstreams + j]);
//        }
//        printf("\n");
//    }
//    printf("Vector:\n");
//    for (int i = 0; i < nstreams; i++) {
//        printf("%f ", vector[i]);
//    }
//    printf("\n");
    printf("Result:\n");
    for (int i = 0; i < nstreams; i++) {
        printf("%f ", result[i]);
    }
    printf("\n");
    
    // Release resources
    for (int i = 0; i < nstreams; i++) {
      hipStreamDestroy(streams[i]);
    }

    free(streams);
    hipEventDestroy(start_event);
    hipEventDestroy(stop_event);

    // Free memory on device
    hipFree(d_matrix);
    hipFree(d_vector);
    hipFree(d_result);

    return 0;
}

